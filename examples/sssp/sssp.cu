#include <cstdlib>  // EXIT_SUCCESS
#include <gunrock/applications/sssp.hxx>
#include "sssp_cpu.hxx"  // Reference implementation

using namespace gunrock;
using namespace memory;

void test_sssp(int num_arguments, char** argument_array) {
  if (num_arguments != 2) {
    std::cerr << "usage: ./bin/<program-name> filename.mtx" << std::endl;
    exit(1);
  }
  
  // >>
  // Enable all-to-all memory access
  int num_gpus = 1;
  hipGetDeviceCount(&num_gpus);
  for(int curr = 0; curr < num_gpus; curr++) {
    hipSetDevice(curr);
    for(int peer = 0; peer < num_gpus; peer++) {
      if(curr == peer) continue;
      hipDeviceEnablePeerAccess(peer, 0);
    }
  }
  // <<

  // --
  // Define types

  using vertex_t = int;
  using edge_t = int;
  using weight_t = float;

  // --
  // IO

  std::string filename = argument_array[1];

  io::matrix_market_t<vertex_t, edge_t, weight_t> mm;

  using csr_t =
      format::csr_t<memory::memory_space_t::device, vertex_t, edge_t, weight_t>;
  csr_t csr;
  csr.from_coo(mm.load(filename));

  // --
  // Build graph

  auto G = graph::build::from_csr<memory_space_t::device, graph::view_t::csr>(
      csr.number_of_rows,               // rows
      csr.number_of_columns,            // columns
      csr.number_of_nonzeros,           // nonzeros
      csr.row_offsets.data().get(),     // row_offsets
      csr.column_indices.data().get(),  // column_indices
      csr.nonzero_values.data().get()   // values
  );  // supports row_indices and column_offsets (default = nullptr)

  // --
  // Params and memory allocation
  
for(int i = 0; i < 100; i++) {
    
  vertex_t single_source = 0;

  vertex_t n_vertices = G.get_number_of_vertices();
  thrust::device_vector<weight_t> distances(n_vertices);
  thrust::device_vector<vertex_t> predecessors(n_vertices);

  // --
  // GPU Run
  
  float gpu_elapsed = gunrock::sssp::run(
      G, single_source, distances.data().get(), predecessors.data().get());

  // --
  // CPU Run

  thrust::host_vector<weight_t> h_distances(n_vertices);
  thrust::host_vector<vertex_t> h_predecessors(n_vertices);

  float cpu_elapsed = sssp_cpu::run<csr_t, vertex_t, edge_t, weight_t>(
      csr, single_source, h_distances.data(), h_predecessors.data());

  int n_errors = sssp_cpu::compute_error(distances, h_distances);

  // --
  // Log + Validate

  std::cout << "GPU Distances (output) = ";
  thrust::copy(distances.begin(), distances.end(),
               std::ostream_iterator<weight_t>(std::cout, " "));
  std::cout << std::endl;

  std::cout << "CPU Distances (output) = ";
  thrust::copy(h_distances.begin(), h_distances.end(),
               std::ostream_iterator<weight_t>(std::cout, " "));
  std::cout << std::endl;

  std::cout << "GPU Elapsed Time : " << gpu_elapsed << " (ms)" << std::endl;
  std::cout << "CPU Elapsed Time : " << cpu_elapsed << " (ms)" << std::endl;
  std::cout << "Number of errors : " << n_errors << std::endl;
  
}

}

int main(int argc, char** argv) {
  test_sssp(argc, argv);
  return EXIT_SUCCESS;
}
